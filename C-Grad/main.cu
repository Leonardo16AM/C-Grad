#include "value.h"
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

int main() {
    
    value a(10.0);
    value b(20.0);
    value d=a+b;

    value c=30;
    value e=d*c;

    e.backpropagate();

    e.print();



    return 0;
}
